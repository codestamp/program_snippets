#include <hip/hip_runtime.h>
#include <iostream>

int main() {
	float **Tab2D;
	int X=5,Y=10;

	Tab2D=new float*[X];

	for(int i=0;i<X;i++) {
		Tab2D[i]=new float[Y];
	}

	//intialize with some random values
	for(int i=0;i<X;i++) {
		for(int j=0;j<Y;j++) {
			Tab2D[i][j]=(float)(rand()%10);
		}
	}

	//print
	for(int i=0;i<X;i++) {
		std::cout<<"\n";
		for(int j=0;j<Y;j++) {
			std::cout << Tab2D[i][j] << ' ';
		}
	}
	
	std::cout << std::endl;
	//free memory
	for(int i=0;i<X;i++)
		delete[] Tab2D[i];

	delete[] Tab2D;

	return 0;
}
