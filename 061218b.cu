#include <hip/hip_runtime.h>
/*
   declaring variables in CUDA kernel
   https://stackoverflow.com/questions/17933702/declaring-variables-in-a-cuda-kernel
   Using -Xptxas=-v option we find that the variable a is not considered in the optimization
*/


__global__ void kernel(float* delt,float* deltb) {
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	float a;
	a=delt[i]+deltb[i];
	a+=1;
}

int main() {
	float *h_delt,*h_deltb;
	h_delt=(float*)malloc(sizeof(float)*400);
	h_deltb=(float*)malloc(sizeof(float)*400);

	for(int i=0;i<400;i++) {
		h_delt[i]=i;
		h_deltb[i]=i;
	}

	float *d_delt,*d_deltb;
	hipMalloc((void**)&d_delt,sizeof(float)*400);
	hipMalloc((void**)&d_deltb,sizeof(float)*400);

	hipMemcpy(d_delt,h_delt,sizeof(float)*400,hipMemcpyHostToDevice);
	hipMemcpy(d_deltb,h_deltb,sizeof(float)*400,hipMemcpyHostToDevice);

	int threads=200;
	uint3 blocks=make_uint3(200,1,1);
	kernel<<<blocks,threads>>>(d_delt,d_deltb);

	free(h_delt); free(h_deltb);
	hipFree(d_delt); hipFree(d_deltb);

	return 0;
}
