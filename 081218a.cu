
#include <hip/hip_runtime.h>
#include <stdio.h>

typedef char mytype;

int main() {
	int rows=10,cols=10;
	mytype **hMat=new mytype*[rows];
	hMat[0]=new mytype[rows*cols];

	for(int i=1;i<rows;i++)
		hMat[i]=hMat[i-1]+cols;

	//initialize 2D arrays
	for(int i=0;i<rows;i++) 
		for(int j=0;j<cols;j++)
			hMat[i][j]=i+j;

	mytype *dArr;
	hipMalloc((void**)&dArr,rows*cols*sizeof(mytype));
	
	//copy to device
	hipMemcpy(dArr,hMat[0],sizeof(mytype)*rows*cols,hipMemcpyHostToDevice);

	//kernel call

	//copy from device
	hipMemcpy(hMat[0],dArr,sizeof(mytype)*rows*cols,hipMemcpyDeviceToHost);

	return 0;
}

	
