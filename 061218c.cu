/*
*	Source: https://stackoverflow.com/questions/24704710/cuda-ptxas-options-v-shared-memory-and-cudafuncattributes-sharedsizebytes-do
* 	Three issues get clarified here:
*	1. use of cudaFuncAttributes(&attr,kernel), refer the documentation at
*	   http://developer.download.nvidia.com/compute/cuda/3_0/toolkit/docs/online/group__CUDART__HIGHLEVEL_g0b85e087210b47056cb6fc03a0e264e8.html
*	   This function obtains the attributes of the function (kernel), kernel must be a global function
*	2. Second is an issue on the sidelines about using printf statement inside kernel
*          https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#formatted-output
*	   printf() can be used in devices with compute capability 2.x or higher, must use -arch=sm_20 and cudaDeviceSynchronize() after the kernel 
*          call inside the host function.
*      	3. usage of -Xptxas=-v for reading the memory usage
	   $ nvcc -Wno-deprecated-gpu-targets  -Xptxas=-v -arch=sm_20 -o 061218c 061218c.cu 
	   ptxas info    : 22 bytes gmem, 16 bytes cmem[14]
	   ptxas info    : Compiling entry function '_Z8mykernelv' for 'sm_20'
	   ptxas info    : Function properties for _Z8mykernelv
	       8 bytes stack frame, 0 bytes spill stores, 0 bytes spill loads
	       ptxas info    : Used 17 registers, 128 bytes smem, 32 bytes cmem[0]

*/



#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(){
  __shared__ int data[32];
  printf("Hello\n");
  for (int i = 0; i < 32; i++)
    printf("data[%d] = %d\n", i, data[i]);
}

int main(){

  hipFuncAttributes attr;
  mykernel<<<1,1>>>();
  hipDeviceSynchronize();
  hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(mykernel));
  printf("shared mem usage: %zu bytes\n", attr.sharedSizeBytes);
  return 0;
}


