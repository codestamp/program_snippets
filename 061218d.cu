
#include <hip/hip_runtime.h>
#include <stdio.h>
/* 
   using printf in kernel, don't forget to call 
   cudaDeviceSynchronize() after kernel call in the host
*/

__global__ void helloCUDA(float f)
{
    printf("Hello thread %d, f=%f\n", threadIdx.x, f);
}

int main()
{
    helloCUDA<<<1, 5>>>(1.2345f);
    hipDeviceSynchronize();
    return 0;
}
